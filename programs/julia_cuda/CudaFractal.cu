#include "hip/hip_runtime.h"
#include <stdio.h>

#include "CudaFractal.h"


static unsigned int *devOutput;

#define CHECK_DISTANCE 50.0f
#define CHECK_DISTANCE_SQUARE (CHECK_DISTANCE * CHECK_DISTANCE)


inline __device__ unsigned int iterFunJulia(float re,float im,float koefRe,float koefIm)
{
	int iter = 0, tempIter = 0;
	float tempRe, tempIm;

	while(
		(re*re+im*im < CHECK_DISTANCE_SQUARE)
		&& 
		(tempIter+=1)<ITERATION_LIMIT)
	{
		iter = tempIter;		

		tempRe = re*re - im*im + koefRe;
		tempIm = 2.0f*re*im + koefIm;
		
		re = tempRe;
		im = tempIm;
	}

	return iter;
}

inline __device__ void getStartEndIndexes(int threadIndex, int outputSize, int totalNumberOfThreads,int &startIndex, int &endIndex)
{
    int slice = (outputSize / totalNumberOfThreads) + 1;
    
    startIndex = threadIndex * slice;
    endIndex = startIndex + slice;
    if(endIndex > outputSize)
    {
        endIndex = outputSize;
    }    
}

inline __device__ void screenIndexToMathCoord(
    int index, 
    int width, int height, 
    float reLowBound, float reHighBound, float imLowBound, float imHighBound, 
    float &re, float &im, float &dRe, float &dIm, int &rowIndex)
{    
    dIm = (imHighBound - imLowBound) / height;
    dRe = (reHighBound - reLowBound) / width;
    
    rowIndex = index % width;
    
    im = imLowBound + (index / width) * dIm;
    re = reLowBound + (rowIndex) * dRe;    
}

__global__ void deviceFractalFunction(unsigned int *output, int width, int height,float koefRe,float koefIm, float bound, int totalNumberOfThreads)
{
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

	int startIndex, endIndex;
	
	int outputSize = width * height;
	
	getStartEndIndexes(threadIndex,outputSize,totalNumberOfThreads,startIndex,endIndex);

	float re, im, dRe, dIm;
	int rowIndex;
	screenIndexToMathCoord(startIndex, width, height, -bound, bound, -bound, 0.0, re, im, dRe, dIm, rowIndex);
	
	for(int i=startIndex;i<endIndex;i++)
	{
		unsigned int iter = iterFunJulia(re,im,koefRe,koefIm);

		output[i]=iter;
		
		if(rowIndex>=width - 1)
		{		    
		    re = -bound;
		    im += dIm;
		    rowIndex=0;
		}
		else
		{
		    re += dRe;
		    rowIndex++;
		}		
	}
}

bool cudaCalculateFractal(
    unsigned int *output,
    int width, int height,
    float koefRe,float koefIm, 
    float bound,
    int numberBlocks,int threadsPerBlock)
{
	hipError_t cudaStatus=hipSuccess;

    int totalNumberOfThreads = numberBlocks * threadsPerBlock;
        
	deviceFractalFunction<<<numberBlocks, threadsPerBlock>>>(
	    devOutput,
	    width,height,
	    koefRe, koefIm,
	    bound,
	    totalNumberOfThreads
	    );

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "cuda error: %s\r\n", hipGetErrorString(cudaStatus));
        return false;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize error: %d\r\n", (int)cudaStatus);
        return false;
    }	
	
        
    cudaStatus = hipMemcpy(output, devOutput, width * height * sizeof(unsigned int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMemcpy error: %d\r\n", (int)cudaStatus);
		return false;
	}
    
	return true;
}

bool cudaInitFractal(unsigned int size)
{
	hipError_t cudaStatus;
    
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice error: %d\r\n", (int)cudaStatus);
        return false;
    }

    cudaStatus = hipMalloc((void**)&devOutput, size * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMalloc error: %d\r\n", (int)cudaStatus);
		return false;
	}
	
	return true;
}

